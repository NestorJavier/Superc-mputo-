#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

void llenaAleatorio(float arreglo[], int n);
void ImprimeArreglo(float arreglo[], float arreglo2[], float arreglo3[], int n);

__global__ void VecAdd(float* A, float* B, float* C, int N){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < N)
        C[i] = A[i] + B[i];
}

int main(void)
{
    int N = 1024;
    size_t size = N * sizeof(float);
    float * h_A = (float*)malloc(size);
    float * h_B = (float*)malloc(size);
    float * h_C = (float*)malloc(size);
    
    //Inicializar vectores de entrada
    llenaAleatorio(h_A, N);
    llenaAleatorio(h_B, N);

    //Allocate vectors in device memory
    float* d_A;
    hipMalloc(&d_A, size);
    float* d_B;
    hipMalloc(&d_B, size);
    float* d_C;
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock -1)/threadsPerBlock;
    VecAdd<<<blocksPerGrid,threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    ImprimeArreglo(h_A, h_B,h_C, N);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);
}

void llenaAleatorio(float arreglo[], int n)
{
    // Declaracion de variables  
    int iSem = time(NULL);
    int sum = 0;
    // Semilla de rand();  
    srand(iSem);  
    int i;
    for(i = 0; i < n; i++)  
    {        
        arreglo[i] = rand()%100;
    }
}


void ImprimeArreglo(float arreglo[], float arreglo2[], float arreglo3[], int n)
{    
    int i;
    for(i = 0; i < n; i++) 
    {        
        printf("%f + %f = %f\n", arreglo[i], arreglo2[i], arreglo3[i]);
    }
}
