#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void matMul(float* A, float* B, float* C, int nFil, int nCol)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;
	int i = idx + idy * nCol;
	if(idx < nCol && idy < nFil)
	{
		float sum = 0;
		for(int k = 0; k < nCol; k++)
		{
			sum += A[idy * nCol + k] * B[k * nCol + idx];
		}
	C[i] = sum;
	}
}

int main()
{
	int nFil = 5;
	int nCol = 5;
	int N = nFil * nCol;
	size_t size = N * sizeof(float);
	
	float* h_A = (float*)malloc(size);
	float* h_B = (float*)malloc(size);
	float* h_C = (float*)malloc(size);

	//Initialialize input vectors

	for(int i = 0; i < nFil; i++)
	{
		for(int j = 0; j < nCol; j++)
		{
			h_A[i * nCol + j] = 1;
			h_B[i * nCol + j] = 2;
		}
	}

	
	float* d_A;
	hipMalloc(&d_A, size);
	float* d_B;
	hipMalloc(&d_B, size);
	float* d_C;
	hipMalloc(&d_C, size);

	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);	


	int threadsPerBlock = 256;
	int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
	matMul<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, nFil, nCol);
	
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	printf("\n\nMatriz resultante:\n");
	for(int i = 0; i < N; i++){
		for(int j = 0; j < N; j++){
			printf("%.2f", h_C[i*nCol+j]);
		}
		printf("\n");	
	}
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	free(h_A);
	free(h_B);
	free(h_C);

	return(0);
}
